#include "hip/hip_runtime.h"
#include "camera.h"
#include <cmath>
#include <cstring>
#include <utility>

Camera::Camera(World* world, Vec3 Pos,int ViewWidth,int ViewHeight,int fromOrigin, float FOV) : CFrame(Quaternion(Pos)), ViewWidth(ViewWidth), ViewHeight(ViewHeight), fromOrigin(fromOrigin), FOV(FOV), View(CameraViewport(this->CFrame, this->ViewWidth, this->ViewHeight, this->FOV, this->fromOrigin)), world(world) {}

Camera::Camera(World* world, Quaternion Coords,int ViewWidth,int ViewHeight,int fromOrigin, float FOV) : CFrame(Coords), ViewWidth(ViewWidth), ViewHeight(ViewHeight), fromOrigin(fromOrigin), FOV(FOV), View(CameraViewport(this->CFrame, this->ViewWidth, this->ViewHeight, this->FOV, this->fromOrigin)), world(world) {}

Quaternion& Camera::move(const Vec3& other) {
    this->CFrame += other;
    this->resetView();
    return this->CFrame;
}

void Camera::setCFrame(Quaternion Pos) {
    this->CFrame = Pos;
    this->resetView();
}

Quaternion& Camera::getCFrame() {
    return this->CFrame;
}

Vec3 Camera::getPos() {
    return this->CFrame.getPos();
}

Quaternion& Camera::rotate(const Vec3& other, double deg) {
    this->CFrame.rotate(other,deg);
    this->resetView();
    return this->CFrame;
}

void Camera::resetView() {
    this->View.SetView(this->CFrame, this->ViewWidth, this->ViewHeight, this->FOV, this->fromOrigin);
}

Vec3 calcRightVector(Quaternion Origin, float FOV, int fromOrigin, int Width, int Height) {
    float imageAspectRatio = Width/Height;
    Vec3 Right = Origin.getRightVector().unitVector();

    return Right * tan(FOV/2) * imageAspectRatio;
}

Vec3 calcUpVector(Quaternion Origin, float FOV, int fromOrigin, int Width, int Height) {
    Vec3 Up = Origin.getUpVector().unitVector();

    return Up * tan(FOV/2);
}

CameraViewport::CameraViewport(Quaternion Origin,int Width, int Height, float FOV, int fromOrigin) : Origin(Origin), Width(Width), Height(Height), unitUp(calcUpVector(Origin,FOV,fromOrigin, Width, Height)), unitRight(calcRightVector(Origin,FOV,fromOrigin, Width, Height)) {}

void CameraViewport::SetView(Quaternion Origin,int Width, int Height, float FOV, int fromOrigin) {
    this->Origin = Origin;
    Vec3 Right = Origin.getRightVector().unitVector();
    Vec3 Up = Origin.getUpVector().unitVector();

    float imageAspectRatio = Width/Height;

    this->unitRight = Right * imageAspectRatio * tan(FOV/2);
    this->unitUp = Up * tan(FOV/2);
}

__global__ void rayTraceKernel(CameraViewport* View, uint32_t* screenBuffer, Triangle* renderable, int RenderableCount, PointLight* lights, int lightCount, Vec3 CFramePos, int MaxDist, int maxBounces, float ambience,float shadow) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int screenWidth = View->Width;
    int screenHeight = View->Height;
    if (index >= screenWidth * screenHeight) return;

    int X = index % screenWidth;
    int Y = index / screenWidth;


    Vec3 newPos = View->Origin.getPos() + View->unitRight * (2*((X+0.5)/screenWidth) - 1) + View->unitUp * (1 - 2*((Y+0.5)/screenHeight)) + View->Origin.getLookVector();
    Ray RenderRay = Ray(View->Origin.getPos() , (newPos - View->Origin.getPos() ).unitVector());
    float Intensity = ambience;

    Triangle* CurrTri = nullptr;
    Triangle* LastTri = nullptr;

    screenBuffer[index] = 0x000000FF;

    for (int v = 0; v < maxBounces; v++) {
        float maxed = MaxDist;
        CurrTri = nullptr;

        for (int i = 0; i < RenderableCount; i++) {
            if (LastTri == &renderable[i]) {
                continue;
            }
            RayIntersectResult Res = renderable[i].rayIntersect(RenderRay);
            if (Res.hit && Res.t < maxed && Res.t > EPSILON) {
                maxed = Res.t;
                CurrTri = &renderable[i];
            }
        }

        LastTri = CurrTri;
        
        if (CurrTri != nullptr) {
            RenderRay = Ray(RenderRay.getPosAtDist(maxed), (RenderRay.getDirection() - CurrTri->getNorm()*2.0f*CurrTri->getNorm().dot(RenderRay.getDirection())).unitVector());
        } else {
            Ray OldRay = RenderRay;

            for (int i = 0; i < lightCount; i++) {
                PointLight light = lights[i];
                RenderRay = Ray(RenderRay.getPos(), (light.Pos-RenderRay.getPos()));

                // Shadow Calculations
                bool NotHit = true;
                for (int i = 0; i < RenderableCount; i++) {
                    if (LastTri == &renderable[i]) {
                        continue;
                    }
                    RayIntersectResult Res = renderable[i].rayIntersect(RenderRay);
                    if (Res.hit && Res.t > EPSILON) {
                        NotHit = false;
                        break;
                    }
                }
                if (NotHit) {
                    Intensity = min(max((light.Pos-OldRay.getPos()).unitVector().dot(OldRay.Dir), 0.0f) + Intensity, 1.0f);
                } else {
                    Intensity = shadow;
                }


            }
            screenBuffer[index] = (CurrTri != nullptr) ? CurrTri->getColor().toUint32( Intensity ) : 0x000000FF;
        }
    }
    
    if (CurrTri != nullptr) {

        Ray OldRay = RenderRay;
        bool NotHit = true;

        for (int i = 0; i < lightCount; i++) {
            PointLight light = lights[i];
            RenderRay = Ray(RenderRay.getPos(), (light.Pos-RenderRay.getPos()));

            // Shadow Calculations
            NotHit = true;
            for (int i = 0; i < RenderableCount; i++) {
                if (LastTri == &renderable[i]) {
                    continue;
                }
                RayIntersectResult Res = renderable[i].rayIntersect(RenderRay);
                if (Res.hit && Res.t > EPSILON) {
                    NotHit = false;
                    break;
                }
            }
            if (NotHit) { //CHANGE LATER
                Intensity = min(max((light.Pos-OldRay.getPos()).unitVector().dot(OldRay.Dir), 0.0f) + Intensity, 1.0f);
            } else {
                Intensity = shadow;
            }


        }
        screenBuffer[index] = (CurrTri != nullptr) ? CurrTri->getColor().toUint32( Intensity ) : 0x000000FF;
    } 
    
}

// Temporary 
#define MAXDIST 200
#define AMBIENT 0.25
#define SHADOW 0.1
#define MAXBOUNCES 1
#define THREADSPERBLOCK 512

void Camera::raytrace(Viewport& screen) {
    CameraViewport View = this->View;
    int bufferSize = View.Width * View.Height;
    uint32_t* screenBuffer;
    hipMalloc(&screenBuffer, bufferSize * sizeof(uint32_t));

    CameraViewport* CamView;
    hipMalloc(&CamView, sizeof(CameraViewport));
    hipMemcpy(CamView, &View, sizeof(CameraViewport), hipMemcpyHostToDevice);

    Triangle* Tris;
    std::pair<const Triangle*,int> TriList = this->world->getTris(View.Origin);
    hipMalloc(&Tris, TriList.second * sizeof(Triangle));
    hipMemcpy(Tris, TriList.first, TriList.second * sizeof(Triangle), hipMemcpyHostToDevice);

    PointLight* Lights;
    std::pair<const PointLight*,int> LightList = this->world->getPointLights(View.Origin);
    hipMalloc(&Lights, LightList.second * sizeof(PointLight));
    hipMemcpy(Lights, LightList.first, LightList.second * sizeof(PointLight), hipMemcpyHostToDevice);

    int threadsPerBlock = THREADSPERBLOCK; 
    int blocks = (bufferSize + threadsPerBlock - 1) / threadsPerBlock;

    rayTraceKernel<<<blocks, threadsPerBlock>>>(CamView, screenBuffer, Tris, TriList.second, Lights, LightList.second, this->CFrame.getPos(), MAXDIST, MAXBOUNCES,AMBIENT, SHADOW);

    hipDeviceSynchronize();

    hipError_t err = hipMemcpy(screen.lockAndGetPixels(),screenBuffer,bufferSize * sizeof(uint32_t),hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cout << "Problem occured: " << err << "\n";
    }

    screen.setDisplay();
    screen.Update();

    hipFree(screenBuffer);
    hipFree(CamView);
    hipFree(Tris);
}

CameraViewport::~CameraViewport() {
}

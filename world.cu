#include "world.h"
#include <utility>

PointLight* getDevicePointLights(std::vector<PointLight> PointLights) {
    PointLight* devicePointLights = nullptr;
    hipMalloc((void**) &devicePointLights, PointLights.size()* sizeof(PointLight));
    hipMemcpy(devicePointLights, PointLights.data(), PointLights.size() * sizeof(PointLight), hipMemcpyHostToDevice);

    return devicePointLights;
}

World::World(std::vector<Object> Renderable, std::vector<PointLight> PointLights) : Tree(BVHTree(Renderable)), Tris(this->Tree.createDeviceTris()), TriIndexes(this->Tree.createDeviceTriIndexes()),
    BVHNodes(this->Tree.createDeviceBVHNodes()), PointLights(getDevicePointLights(PointLights)), LightCount(PointLights.size()) {}

World::~World() {
    std::cout << "FREED WORLD\n";
    hipFree(this->BVHNodes);
    hipFree(this->Tris);
    hipFree(this->TriIndexes);
}